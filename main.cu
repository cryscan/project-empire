#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <hip/hip_runtime.h>
#include <>

#include "common.cuh"
#include "heap.cuh"
#include "hashtable.cuh"
#include "astar.cuh"

struct SlidingPad {
    using Node = uint64_t;
    using Value = unsigned int;
    using Heap = Heap<Node, Value>;
    using State = State<Node, Value>;
    using StatePtr = Arc<State>;
    using Hashtable = Hashtable<Node, Value>;

    enum Direction {
        UP = 0,
        RIGHT,
        DOWN,
        LEFT,
    };

    static __device__ Value heuristic(Node s, Node t) {
        Node filter = 0xf;
        Value result = 0;
        for (int i = 0; i < 16; ++i, filter <<= 4) {
            auto x = (s & filter) >> (4 * i);
            auto y = (t & filter) >> (4 * i);
            result += x > y ? x - y : y - x;
        }
        return result;
    }

    static __device__ StatePtr expand_direction(const StatePtr& state, Node t, Direction direction) {
        /*  Board
         *      0   1   2   3
         *      4   5   6   7
         *      8   9   10  11
         *      12  13  14  15
         *
         *  Node
         *      15  14  13  12  11  10  9   8   7   6   5   4   3   2   1   0
         */
        State current = *state;
        Node filter = 0xf;
        int x, y;
        for (int i = 0; i < 16; ++i, filter <<= 4) {
            if ((current.node & filter) == 0) {
                x = i / 4;
                y = i % 4;
                break;
            }
        }

        if (direction == UP && x > 0) {
            auto selected = current.node & (filter >> 16);
            State next;
            next.node = (current.node | (selected << 16)) ^ selected;
            next.g = current.g + 1;
            // next.f = next.g + heuristic(next.node, t);
            next.prev = state;
            return make_arc<State>(next);
        }

        if (direction == DOWN && x < 3) {
            auto selected = current.node & (filter << 16);
            State next;
            next.node = (current.node | (selected >> 16)) ^ selected;
            next.g = current.g + 1;
            // next.f = next.g + heuristic(next.node, t);
            next.prev = state;
            return make_arc<State>(next);
        }

        if (direction == LEFT && y > 0) {
            auto selected = current.node & (filter >> 4);
            State next;
            next.node = (current.node | (selected << 4)) ^ selected;
            next.g = current.g + 1;
            // next.f = next.g + heuristic(next.node, t);
            next.prev = state;
            return make_arc<State>(next);
        }

        if (direction == RIGHT && y < 3) {
            auto selected = current.node & (filter << 4);
            State next;
            next.node = (current.node | (selected >> 4)) ^ selected;
            next.g = current.g + 1;
            // next.f = next.g + heuristic(next.node, t);
            next.prev = state;
            return make_arc<State>(next);
        }

        return {};
    }

    static __device__ void expand(StatePtr* s_dev, const StatePtr& state, Node t) {
        auto index = blockIdx.x * blockDim.x + threadIdx.x;
        for (auto d: {UP, RIGHT, DOWN, LEFT}) {
            s_dev[index * max_expansion + d] = expand_direction(state, t, d);
        }
    }
};

/*
__global__ void test(Heap* heap_dev, unsigned* buf) {
    auto s1 = make_arc<State>();
    auto s2 = make_arc<State>();
    auto s3 = make_arc<State>();
    auto s4 = make_arc<State>();
    auto s5 = make_arc<State>();

    s1->f = 2;
    s2->f = 1;
    s3->f = 5;
    s4->f = 4;
    s5->f = 2;

    heap_dev->push(s1);
    heap_dev->push(s2);
    heap_dev->push(s3);
    heap_dev->push(s4);
    heap_dev->push(s5);

    buf[0] = heap_dev->pop()->f;
    buf[1] = heap_dev->pop()->f;
    buf[2] = heap_dev->pop()->f;
    buf[3] = heap_dev->pop()->f;
    buf[4] = heap_dev->pop()->f;
}

__global__ void test_hash(HashtableType* table_dev) {
    size_t index = threadIdx.x + blockIdx.x * blockDim.x;

    auto key = index;
    auto value = make_arc<State>();
    value->node = key;
    table_dev->insert(key, value);
}

__global__ void test_hash_find(HashtableType* table_dev, uint64_t* buf_dev, bool* bool_dev) {
    size_t index = threadIdx.x + blockIdx.x * blockDim.x;
    auto key = index;

    Arc<State> result;
    if (index == 42) key = 1000;
    bool_dev[index] = table_dev->find(key, result);
    if (result) buf_dev[index] = result->node;
}
 */

template<typename Game>
__global__ void init_heaps(typename Game::Heap* heaps_dev, typename Game::Node s, typename Game::Node t) {
    typename Game::State state;
    state.node = s;
    state.g = 0;
    state.f = Game::heuristic(s, t);
    heaps_dev[0].push(make_arc<typename Game::State>(state));
}

template<typename Game>
__global__ void extract_nodes(Arc<typename Game::State>* s_dev, typename Game::Node* nodes_dev) {
    auto index = blockIdx.x * blockDim.x + threadIdx.x;
    nodes_dev[index] = s_dev[index] ? s_dev[index]->node : 0;
}

int main(int argc, char** argv) {
    using Game = SlidingPad;

    std::vector<Game::Heap> heaps(num_heaps);

    Game::Heap* heaps_dev;
    HANDLE_RESULT(hipMalloc(&heaps_dev, num_heaps * sizeof(Game::Heap)))
    HANDLE_RESULT(hipMemcpy(heaps_dev, heaps.data(), num_heaps * sizeof(Game::Heap), hipMemcpyHostToDevice))

    Game::StatePtr* s_dev;
    HANDLE_RESULT(hipMalloc(&s_dev, num_expanded_states * sizeof(Game::StatePtr)))
    HANDLE_RESULT(hipMemset(s_dev, 0, num_expanded_states * sizeof(Game::StatePtr)))

    Game::StatePtr* m_dev;
    HANDLE_RESULT(hipMalloc(&m_dev, sizeof(Game::StatePtr)))

    bool found;
    bool* found_dev;
    HANDLE_RESULT(hipMalloc(&found_dev, sizeof(bool)))

    Game::Node s = 0xfedcba9876543210;
    Game::Node t = 0x0123456789abcdef;

    init_heaps<Game><<<1, 1>>>(heaps_dev, s, t);
    extract_expand<Game><<<1, num_heaps, num_heaps * sizeof(Game::StatePtr)>>>(
            heaps_dev,
            s_dev,
            m_dev,
            t);
    compare_heap_best<Game><<<1, num_heaps, num_heaps * sizeof(Game::StatePtr)>>>(heaps_dev, m_dev, found_dev);

    HANDLE_RESULT(hipMemcpy(&found, found_dev, sizeof(bool), hipMemcpyDeviceToHost))

    Game::Node nodes_cpu[num_expanded_states];
    Game::Node* nodes_dev;
    HANDLE_RESULT(hipMalloc(&nodes_dev, num_expanded_states * sizeof(Game::Node)))

    // extract nodes from pointers
    extract_nodes<Game><<<max_expansion, num_heaps>>>(s_dev, nodes_dev);

    HANDLE_RESULT(
            hipMemcpy(nodes_cpu, nodes_dev, num_expanded_states * sizeof(Game::Node), hipMemcpyDeviceToHost))

    HANDLE_RESULT(hipFree(heaps_dev))
    HANDLE_RESULT(hipFree(s_dev))
    HANDLE_RESULT(hipFree(m_dev))
    HANDLE_RESULT(hipFree(found_dev))
    HANDLE_RESULT(hipFree(nodes_dev))

    // test <<< 1, 1 >>>(heap_dev, buf_dev);

    /*
    constexpr size_t thread_count = 1024;
    constexpr size_t table_size = 1024 * 1024;

    HashtableType table(table_size);

    HashtableType* table_dev;
    HANDLE_RESULT(hipMalloc(&table_dev, sizeof(HashtableType)))
    HANDLE_RESULT(hipMemcpy(table_dev, &table, sizeof(HashtableType), hipMemcpyHostToDevice))

    uint64_t* buf_dev;
    HANDLE_RESULT(hipMalloc(&buf_dev, thread_count * sizeof(uint64_t)));

    bool* bool_dev;
    HANDLE_RESULT(hipMalloc(&bool_dev, thread_count * sizeof(bool)));

    test_hash<<<1, thread_count>>>(table_dev);

    hipDeviceSynchronize();

    test_hash_find<<<1, thread_count>>>(table_dev, buf_dev, bool_dev);

    uint64_t buf[thread_count];
    HANDLE_RESULT(hipMemcpy(buf, buf_dev, thread_count * sizeof(uint64_t), hipMemcpyDeviceToHost))

    bool bool_buf[thread_count];
    HANDLE_RESULT(hipMemcpy(bool_buf, bool_dev, thread_count * sizeof(bool), hipMemcpyDeviceToHost))

    std::cout << "elements: \n";
    for (auto element: buf) {
        std::cout << element << '\n';
    }

    std::cout << "finds: \n";
    for (auto element: bool_buf) {
        std::cout << element << '\n';
    }*/


    /*constexpr uint64_t HEAP_CAPACITY = 1024;
    Heap h(HEAP_CAPACITY);
    Heap* h_dev;
    HANDLE_RESULT(hipMalloc(&h_dev, sizeof(Heap)))
    HANDLE_RESULT(hipMemcpy(h_dev, &h, sizeof(Heap), hipMemcpyHostToDevice))


    uint64_t nodesInS[Directions::Direction::NUM_DIRECTIONS];
    uint64_t* nodesInS_dev;
    HANDLE_RESULT(hipMalloc(&nodesInS_dev, Directions::Direction::NUM_DIRECTIONS * sizeof(uint64_t)))

    unsigned valueInDest;
    unsigned* valueInDest_dev;
    HANDLE_RESULT(hipMalloc(&valueInDest_dev, sizeof(unsigned)));*/

    return 0;
}
