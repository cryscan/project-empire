#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>

#include "common.cuh"
#include "heap.cuh"
#include "hashtable.cuh"
#include "astar.cuh"

struct SlidingPad {
    using Node = uint64_t;
    using Value = unsigned int;
    using Heap = Heap<Node, Value>;
    using State = State<Node, Value>;
    using StatePtr = Arc<State>;
    using SerializedState = SerializedState<Node, Value>;
    using Hashtable = Hashtable<Node, Value>;

    enum Direction {
        UP = 0,
        RIGHT = 1,
        DOWN = 2,
        LEFT = 3,
    };

    static __device__ Value heuristic(Node s, Node t) {
        Node filter = 0xf;
        Value result = 0;
        for (int i = 0; i < 16; ++i, filter <<= 4) {
            auto x = (s & filter) >> (4 * i);
            auto y = (t & filter) >> (4 * i);
            if (x != y && x != 0) ++result;
        }
        return result;
    }

    static __device__ StatePtr expand_direction(const StatePtr& state, Direction direction) {
        /*  Board
         *      0   1   2   3
         *      4   5   6   7
         *      8   9   10  11
         *      12  13  14  15
         *
         *  Node
         *      15  14  13  12  11  10  9   8   7   6   5   4   3   2   1   0
         */
        State current = *state;
        Node filter = 0xf;
        int x = -1, y = -1;
        for (int i = 0; i < 16; ++i, filter <<= 4) {
            if ((current.node & filter) == 0) {
                x = i / 4;
                y = i % 4;
                break;
            }
        }

        assert(x >= 0 && x <= 3 && y >= 0 && y <= 3);

        if (direction == UP && x > 0) {
            auto selected = current.node & (filter >> 16);
            State next;
            next.node = (current.node | (selected << 16)) ^ selected;
            next.g = current.g + 1;
            next.prev = state;
            return make_arc<State>(next);
        }

        if (direction == RIGHT && y < 3) {
            auto selected = current.node & (filter << 4);
            State next;
            next.node = (current.node | (selected >> 4)) ^ selected;
            next.g = current.g + 1;
            next.prev = state;
            return make_arc<State>(next);
        }

        if (direction == DOWN && x < 3) {
            auto selected = current.node & (filter << 16);
            State next;
            next.node = (current.node | (selected >> 16)) ^ selected;
            next.g = current.g + 1;
            next.prev = state;
            return make_arc<State>(next);
        }

        if (direction == LEFT && y > 0) {
            auto selected = current.node & (filter >> 4);
            State next;
            next.node = (current.node | (selected << 4)) ^ selected;
            next.g = current.g + 1;
            next.prev = state;
            return make_arc<State>(next);
        }

        return nullptr;
    }

    static __device__ void expand(StatePtr* s_dev, const StatePtr& state) {
        auto index = blockIdx.x * blockDim.x + threadIdx.x;
        for (auto d: {UP, RIGHT, DOWN, LEFT}) {
            s_dev[index * max_expansion + d] = expand_direction(state, d);
        }
    }
};

/*
__global__ void test(Heap* heap_dev, unsigned* buf) {
    auto s1 = make_arc<State>();
    auto s2 = make_arc<State>();
    auto s3 = make_arc<State>();
    auto s4 = make_arc<State>();
    auto s5 = make_arc<State>();

    s1->f = 2;
    s2->f = 1;
    s3->f = 5;
    s4->f = 4;
    s5->f = 2;

    heap_dev->push(s1);
    heap_dev->push(s2);
    heap_dev->push(s3);
    heap_dev->push(s4);
    heap_dev->push(s5);

    buf[0] = heap_dev->pop()->f;
    buf[1] = heap_dev->pop()->f;
    buf[2] = heap_dev->pop()->f;
    buf[3] = heap_dev->pop()->f;
    buf[4] = heap_dev->pop()->f;
}

__global__ void test_hash(HashtableType* table_dev) {
    size_t index = threadIdx.x + blockIdx.x * blockDim.x;

    auto key = index;
    auto value = make_arc<State>();
    value->node = key;
    table_dev->insert(key, value);
}

__global__ void test_hash_find(HashtableType* table_dev, uint64_t* buf_dev, bool* bool_dev) {
    size_t index = threadIdx.x + blockIdx.x * blockDim.x;
    auto key = index;

    Arc<State> result;
    if (index == 42) key = 1000;
    bool_dev[index] = table_dev->find(key, result);
    if (result) buf_dev[index] = result->node;
}
 */

template<typename Game>
__global__ void extract_states(typename Game::StatePtr* s_dev, typename Game::SerializedState* states_dev) {
    auto index = blockIdx.x * blockDim.x + threadIdx.x;
    if (auto& ptr = s_dev[index]) {
        states_dev[index] = typename Game::SerializedState(*ptr);
    }
}

template<typename Game>
__global__ void extract_heap_bests(typename Game::Heap* heaps_dev, typename Game::SerializedState* states_dev) {
    auto index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= num_heaps) return;

    auto& heap = heaps_dev[index];
    if (auto& ptr = heap.data()[0]) {
        auto state = *ptr;
        states_dev[index] = typename Game::SerializedState(state);
    }
}

template<typename Game>
__global__ void extract_chain(typename Game::StatePtr* m_dev, typename Game::SerializedState* solution_dev) {
    auto ptr = *m_dev;
    for (auto i = 0u; ptr; ++i) {
        auto state = *ptr;
        solution_dev[i] = typename Game::SerializedState(state);
        ptr = state.prev;
    }
}

int main(int argc, char** argv) {
    using Game = SlidingPad;

    std::vector<Game::Heap> heaps(num_heaps);
    Game::Hashtable hashtable;

    Game::Heap* heaps_dev;
    HANDLE_RESULT(hipMalloc(&heaps_dev, num_heaps * sizeof(Game::Heap)))
    HANDLE_RESULT(hipMemcpy(heaps_dev, heaps.data(), num_heaps * sizeof(Game::Heap), hipMemcpyHostToDevice))

    Game::Hashtable* hashtable_dev;
    HANDLE_RESULT(hipMalloc(&hashtable_dev, sizeof(Game::Hashtable)))
    HANDLE_RESULT(hipMemcpy(hashtable_dev, &hashtable, sizeof(Game::Hashtable), hipMemcpyHostToDevice))

    Game::StatePtr* s_dev;
    HANDLE_RESULT(hipMalloc(&s_dev, num_expanded_states * sizeof(Game::StatePtr)))
    HANDLE_RESULT(hipMemset(s_dev, 0, num_expanded_states * sizeof(Game::StatePtr)))

    Game::StatePtr* t_dev;
    HANDLE_RESULT(hipMalloc(&t_dev, num_expanded_states * sizeof(Game::StatePtr)))
    HANDLE_RESULT(hipMemset(t_dev, 0, num_expanded_states * sizeof(Game::StatePtr)))

    Game::StatePtr* m_dev;
    HANDLE_RESULT(hipMalloc(&m_dev, sizeof(Game::StatePtr)))
    HANDLE_RESULT(hipMemset(m_dev, 0, sizeof(Game::StatePtr)))

    bool found;
    bool* found_dev;
    HANDLE_RESULT(hipMalloc(&found_dev, sizeof(bool)))
    HANDLE_RESULT(hipMemset(found_dev, 0, sizeof(bool)))

    Game::Node start = 0xFEDCBA9876543210;
    // Game::Node target = 0xEFDCBA8976543210;
    // Game::Node target = 0xFAEDB95C76803214;
    Game::Node target = 0xFAEDB9C478513620;

    /*
    Game::Node* start_dev;
    HANDLE_RESULT(hipMalloc(&start_dev, sizeof(Game::Node)))
    HANDLE_RESULT(hipMemcpy(start_dev, &start, sizeof(Game::Node), hipMemcpyHostToDevice))

    Game::Node* target_dev;
    HANDLE_RESULT(hipMalloc(&target_dev, sizeof(Game::Node)))
    HANDLE_RESULT(hipMemcpy(target_dev, &target, sizeof(Game::Node), hipMemcpyHostToDevice))
     */

    init_heaps<Game><<<1, 1>>>(heaps_dev, start, target);
    HANDLE_RESULT(hipGetLastError())

    for (int i = 0; i < solution_size; ++i) {
        std::cout << "Iteration " << i << '\n';

        extract_expand<Game><<<1, num_heaps, num_heaps * sizeof(Game::StatePtr)>>>(
                heaps_dev,
                s_dev,
                m_dev,
                target);
        HANDLE_RESULT(hipGetLastError())

        compare_heap_best<Game><<<1, num_heaps, num_heaps * sizeof(Game::StatePtr)>>>(
                heaps_dev,
                m_dev,
                found_dev);
        HANDLE_RESULT(hipGetLastError())

        // if (i % 16 == 0) {
        HANDLE_RESULT(hipMemcpy(&found, found_dev, sizeof(bool), hipMemcpyDeviceToHost))
        if (found) break;
        // }

        remove_duplication<Game><<<max_expansion, num_heaps>>>(hashtable_dev, s_dev, t_dev);
        HANDLE_RESULT(hipGetLastError())

        reinsert<Game><<<1, num_heaps>>>(hashtable_dev, heaps_dev, t_dev, target);
        HANDLE_RESULT(hipGetLastError())
    }

    HANDLE_RESULT(hipDeviceSynchronize())

    Game::SerializedState solution[solution_size];
    Game::SerializedState* solution_dev;
    HANDLE_RESULT(hipMalloc(&solution_dev, solution_size * sizeof(Game::SerializedState)))
    HANDLE_RESULT(hipMemset(solution_dev, 0, solution_size * sizeof(Game::SerializedState)))

    extract_chain<Game><<<1, 1>>>(m_dev, solution_dev);

    HANDLE_RESULT(hipMemcpy(solution,
                             solution_dev,
                             solution_size * sizeof(Game::SerializedState),
                             hipMemcpyDeviceToHost))

    std::cout << std::endl << "Solution:\n";
    for (auto x: solution) {
        if (x.node == 0) break;
        std::cout << x.g << ' ' << x.f << std::endl;

        Game::Node filter = 0xf;
        for (auto i = 0u; i < 4; ++i) {
            for (auto j = 0u; j < 4; ++j) {
                auto n = (filter & x.node) >> ((4 * i + j) * 4);
                std::cout << n << '\t';
                filter <<= 4;
            }
            std::cout << '\n';
        }
        std::cout << std::endl;
    }

    Game::SerializedState heap_bests[num_heaps];
    Game::SerializedState* heap_bests_dev;
    HANDLE_RESULT(hipMalloc(&heap_bests_dev, num_heaps * sizeof(Game::SerializedState)))

    extract_heap_bests<Game><<<1, num_heaps>>>(heaps_dev, heap_bests_dev);

    HANDLE_RESULT(hipMemcpy(heap_bests,
                             heap_bests_dev,
                             num_heaps * sizeof(Game::SerializedState),
                             hipMemcpyDeviceToHost))

    {
        std::ofstream fs("best.txt");
        auto k = 0u;
        fs << "Bests:\n";
        for (auto x: heap_bests) {
            if (x.node == 0) continue;
            fs << k << ' ' << x.g << ' ' << x.f << std::endl;

            Game::Node filter = 0xf;
            for (auto i = 0u; i < 4; ++i) {
                for (auto j = 0u; j < 4; ++j) {
                    auto n = (filter & x.node) >> ((4 * i + j) * 4);
                    fs << n << '\t';
                    filter <<= 4;
                }
                fs << '\n';
            }
            fs << std::endl;
            ++k;
        }
    }

    /*
    std::vector<Game::SerializedState> t_states(num_expanded_states);
    Game::SerializedState* t_states_dev;
    HANDLE_RESULT(hipMalloc(&t_states_dev, num_expanded_states * sizeof(Game::SerializedState)))

    extract_states<Game><<<max_expansion, num_heaps>>>(t_dev, t_states_dev);

    HANDLE_RESULT(hipMemcpy(
            t_states.data(),
            t_states_dev,
            num_expanded_states * sizeof(Game::SerializedState),
            hipMemcpyDeviceToHost))

    {
        std::ofstream fs("t.txt");
        auto k = 0u;
        fs << "T states:\n";
        for (auto x: t_states) {
            if (x.node == 0) continue;
            fs << k << ' ' << x.g << ' ' << x.f << std::endl;

            Game::Node filter = 0xf;
            for (auto i = 0u; i < 4; ++i) {
                for (auto j = 0u; j < 4; ++j) {
                    auto n = (filter & x.node) >> ((4 * i + j) * 4);
                    fs << n << '\t';
                    filter <<= 4;
                }
                fs << '\n';
            }
            fs << std::endl;
            ++k;
        }
    }

    std::vector<Game::SerializedState> s_states(num_expanded_states);
    Game::SerializedState* s_states_dev;
    HANDLE_RESULT(hipMalloc(&s_states_dev, num_expanded_states * sizeof(Game::SerializedState)))

    // extract nodes from pointers
    extract_states<Game><<<max_expansion, num_heaps>>>(s_dev, s_states_dev);
    extract_states<Game><<<max_expansion, num_heaps>>>(t_dev, t_states_dev);

    HANDLE_RESULT(hipMemcpy(
            s_states.data(),
            s_states_dev,
            num_expanded_states * sizeof(Game::SerializedState),
            hipMemcpyDeviceToHost))

    // HANDLE_RESULT(hipFree(heaps_dev))
    // HANDLE_RESULT(hipFree(s_dev))
    // HANDLE_RESULT(hipFree(t_dev))
    // HANDLE_RESULT(hipFree(m_dev))
    // HANDLE_RESULT(hipFree(found_dev))

    // test <<< 1, 1 >>>(heap_dev, buf_dev);

    constexpr size_t thread_count = 1024;
    constexpr size_t table_size = 1024 * 1024;

    HashtableType table(table_size);

    HashtableType* table_dev;
    HANDLE_RESULT(hipMalloc(&table_dev, sizeof(HashtableType)))
    HANDLE_RESULT(hipMemcpy(table_dev, &table, sizeof(HashtableType), hipMemcpyHostToDevice))

    uint64_t* buf_dev;
    HANDLE_RESULT(hipMalloc(&buf_dev, thread_count * sizeof(uint64_t)));

    bool* bool_dev;
    HANDLE_RESULT(hipMalloc(&bool_dev, thread_count * sizeof(bool)));

    test_hash<<<1, thread_count>>>(table_dev);

    hipDeviceSynchronize();

    test_hash_find<<<1, thread_count>>>(table_dev, buf_dev, bool_dev);

    uint64_t buf[thread_count];
    HANDLE_RESULT(hipMemcpy(buf, buf_dev, thread_count * sizeof(uint64_t), hipMemcpyDeviceToHost))

    bool bool_buf[thread_count];
    HANDLE_RESULT(hipMemcpy(bool_buf, bool_dev, thread_count * sizeof(bool), hipMemcpyDeviceToHost))

    std::cout << "elements: \n";
    for (auto element: buf) {
        std::cout << element << '\n';
    }

    std::cout << "finds: \n";
    for (auto element: bool_buf) {
        std::cout << element << '\n';
    }

    constexpr uint64_t HEAP_CAPACITY = 1024;
    Heap h(HEAP_CAPACITY);
    Heap* h_dev;
    HANDLE_RESULT(hipMalloc(&h_dev, sizeof(Heap)))
    HANDLE_RESULT(hipMemcpy(h_dev, &h, sizeof(Heap), hipMemcpyHostToDevice))


    uint64_t nodesInS[Directions::Direction::NUM_DIRECTIONS];
    uint64_t* nodesInS_dev;
    HANDLE_RESULT(hipMalloc(&nodesInS_dev, Directions::Direction::NUM_DIRECTIONS * sizeof(uint64_t)))

    unsigned valueInDest;
    unsigned* valueInDest_dev;
    HANDLE_RESULT(hipMalloc(&valueInDest_dev, sizeof(unsigned)));
     */

    return 0;
}
